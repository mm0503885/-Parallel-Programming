/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void printfinal (void);


float *g_value, *gold_value, *g_new_value;
int size = MAXPOINTS+2;
int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */
	   
	   
	   
__global__ void gpu_init_old_value(float *a, float *b, float *c, int i)
{
        int j=blockIdx.x*blockDim.x+threadIdx.x, m=gridDim.x*blockDim.x;
        for(int k=j; k<i; k+=m)
		{
		  a[k] = b[k];
        }
        __syncthreads();
}
__global__ void gpu_update_point(float *a, float *b, float *c, int p, int nsteps)
{
                int i=blockIdx.x*blockDim.x+threadIdx.x;
                float aval = a[i], bval = b[i];
                float cval;
                if (i < p) 
				{
                  for (int j = 0;j<nsteps;j++)
				  {
                    if ((i== 0) || (i  == p - 1))
                        cval = 0.0;
                    else
                        cval = (2.0 * bval) - aval + (0.09 * (-2.0)*bval);
                        aval = bval;
                        bval = cval;
                        __syncthreads();
                  }
                }
                b[i] = bval;
}

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   } 

   hipMemcpy(g_value, values, size, hipMemcpyHostToDevice);
   hipMemcpy(gold_value, oldval, size, hipMemcpyHostToDevice);
   hipMemcpy(g_new_value, newval, size, hipMemcpyHostToDevice);
   gpu_init_old_value<<<30,512>>>(gold_value, g_value, g_new_value, tpoints);
   printf("Updating all points for all time steps...\n");
   gpu_update_point<<<(tpoints/512 + 1),512>>>(gold_value, g_value, g_new_value, tpoints, nsteps);
   hipMemcpy(values, g_value, size, hipMemcpyDeviceToHost);
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
void do_math(int i)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}


/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
	hipMalloc((void**)&g_value, size);
    hipMalloc((void**)&gold_value, size);
    hipMalloc((void**)&g_new_value, size);
	printf("Initializing points on the line...\n");
	init_line();
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}